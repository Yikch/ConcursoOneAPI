#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"

#define NTHREADS 32
#define TILE_DIM 5

__global__ void gpu_canny(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	__shared__ float tile [ TILE_DIM ] [ TILE_DIM+1 ]; 
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
}
__global__ void gpu_hough(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, 
	float *sin_table, float *cos_table)
{

}

void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table,
	int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho, theta, ii, jj;
	uint32_t max;

	for(rho=0;rho<accu_height;rho++)
	{
		for(theta=0;theta<accu_width;theta++)  
		{  

			if(accumulators[(rho*accu_width) + theta] >= threshold)  
			{  
				//Is this point a local maxima (9x9)  
				max = accumulators[(rho*accu_width) + theta]; 
				for(int ii=-4;ii<=4;ii++)  
				{  
					for(int jj=-4;jj<=4;jj++)  
					{  
						if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) )  
						{  
							if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max )  
							{
								max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
							}  
						}  
					}  
				}  

				if(max == accumulators[(rho*accu_width) + theta]) //local maxima
				{
					int x1, y1, x2, y2;  
					x1 = y1 = x2 = y2 = 0;  

					if(theta >= 45 && theta <= 135)  
					{
						if (theta>90) {
							//y = (r - x cos(t)) / sin(t)  
							x1 = width/2;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						} else {
							//y = (r - x cos(t)) / sin(t)  
							x1 = 0;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width*2/5;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					} else {
						//x = (r - y sin(t)) / cos(t);  
						y1 = 0;  
						x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2 = height;  
						x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}
					x1_lines[*lines] = x1;
					y1_lines[*lines] = y1;
					x2_lines[*lines] = x2;
					y2_lines[*lines] = y2;
					(*lines)++;
				}
			}
		}
	}
}


void line_asist_GPU(uint8_t *im, int height, int width,
	uint8_t *imEdge, float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float *sin_table, float *cos_table, 
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *x2, int *y1, int *y2, int *nlines)
{
	int threshold;

	/* CUDA vesion */

	dim3 dimBlock(NTHREADS,NTHREADS);
	int blocks = (height*width/25)/NTHREADS;
	if ((height*width/25)%NTHREADS>0) blocks++;
	dim3 dimGrid(blocks, blocks);

	gpu_canny<<<dimGrid,dimBlock>>>(im, imEdge, NR, G, phi, Gx, Gy, pedge,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();
	
	getlines(threshold, accum, accu_width, accu_height, width, height, 
		sin_table, cos_table,
		x1, y1, x2, y2, nlines);
}
