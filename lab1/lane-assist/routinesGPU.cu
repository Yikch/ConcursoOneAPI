#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"

#define NTHREADS 25
#define TILE_DIM 29  //TILE_DIM = NTHREADS + 4

__global__ void gpu_canny_nr(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	__shared__ float tile [ TILE_DIM ] [ TILE_DIM ]; 
	i = blockIdx.y * NTHREADS + threadIdx.y; 
	j = blockIdx.x * NTHREADS + threadIdx.x; 
	if(i<height && j<width){
		tile[threadIdx.y][threadIdx.x] = im[i*width+j];
		__syncthreads();
		if(i>=2 && i<height-2 && threadIdx.x>=2 && threadIdx.x<TILE_DIM-2 && j >=2 && j<width-2 && threadIdx.y>=2 && threadIdx.y<TILE_DIM-2){
			// Noise reduction
			NR[i*width+j] =
				(2.0*tile[threadIdx.y-2][threadIdx.x-2]	 +  4.0*tile[threadIdx.y-2][threadIdx.x-1] +  5.0*tile[threadIdx.y-2][threadIdx.x] +  4.0*tile[threadIdx.y-2][threadIdx.x+1] + 2.0*tile[threadIdx.y-2][threadIdx.x+2]
				+ 4.0*tile[threadIdx.y-1][threadIdx.x-2] +  9.0*tile[threadIdx.y-1][threadIdx.x-1] + 12.0*tile[threadIdx.y-1][threadIdx.x] +  9.0*tile[threadIdx.y-1][threadIdx.x+1] + 4.0*tile[threadIdx.y-1][threadIdx.x+2]
				+ 5.0*tile[threadIdx.y  ][threadIdx.x-2] + 12.0*tile[threadIdx.y  ][threadIdx.x-1] + 15.0*tile[threadIdx.y  ][threadIdx.x] + 12.0*tile[threadIdx.y  ][threadIdx.x+1] + 5.0*tile[threadIdx.y][threadIdx.x+2]
				+ 4.0*tile[threadIdx.y+1][threadIdx.x-2] +  9.0*tile[threadIdx.y+1][threadIdx.x-1] + 12.0*tile[threadIdx.y+1][threadIdx.x] +  9.0*tile[threadIdx.y+1][threadIdx.x+1] + 4.0*tile[threadIdx.y+1][threadIdx.x+2]
				+ 2.0*tile[threadIdx.y+2][threadIdx.x-2] +  4.0*tile[threadIdx.y+2][threadIdx.x-1] +  5.0*tile[threadIdx.y+2][threadIdx.x] +  4.0*tile[threadIdx.y+2][threadIdx.x+1] + 2.0*tile[threadIdx.y+2][threadIdx.x+2])
				/159.0;
		}
	}

}

__global__ void gpu_canny_gradient(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	__shared__ float tile [ TILE_DIM ] [ TILE_DIM ]; 
	float PI = 3.141593;
	i = blockIdx.y * NTHREADS + threadIdx.y; 
	j = blockIdx.x * NTHREADS + threadIdx.x; 
	if(i<height && j<width){
		tile[threadIdx.y][threadIdx.x] = NR[i*width+j];
		__syncthreads();
		if(i>=2 && i<height-2 && threadIdx.x>=2 && threadIdx.x<TILE_DIM-2 && j >=2 && j<width-2 && threadIdx.y>=2 && threadIdx.y<TILE_DIM-2){
			// Intensity gradient of the image
			Gx[i*width+j] = 
				 (1.0*tile[threadIdx.y-2][threadIdx.x-2] +  2.0*tile[threadIdx.y-2][threadIdx.x-1] +  (-2.0)*tile[threadIdx.y-2][threadIdx.x+1] + (-1.0)*tile[threadIdx.y-2][threadIdx.x+2]
				+ 4.0*tile[threadIdx.y-1][threadIdx.x-2] +  8.0*tile[threadIdx.y-1][threadIdx.x-1] +  (-8.0)*tile[threadIdx.y-1][threadIdx.x+1] + (-4.0)*tile[threadIdx.y-1][threadIdx.x+2]
				+ 6.0*tile[threadIdx.y  ][threadIdx.x-2] + 12.0*tile[threadIdx.y  ][threadIdx.x-1] + (-12.0)*tile[threadIdx.y  ][threadIdx.x+1] + (-6.0)*tile[threadIdx.y  ][threadIdx.x+2]
				+ 4.0*tile[threadIdx.y+1][threadIdx.x-2] +  8.0*tile[threadIdx.y+1][threadIdx.x-1] +  (-8.0)*tile[threadIdx.y+1][threadIdx.x+1] + (-4.0)*tile[threadIdx.y+1][threadIdx.x+2]
				+ 1.0*tile[threadIdx.y+2][threadIdx.x-2] +  2.0*tile[threadIdx.y+2][threadIdx.x-1] +  (-2.0)*tile[threadIdx.y+2][threadIdx.x+1] + (-1.0)*tile[threadIdx.y+2][threadIdx.x+2]);


			Gy[i*width+j] = 
				 ((-1.0)*tile[threadIdx.y-2][threadIdx.x-2] + (-4.0)*tile[threadIdx.y-2][threadIdx.x-1] +  (-6.0)*tile[threadIdx.y-2][threadIdx.x] + (-4.0)*tile[threadIdx.y-2][threadIdx.x+1] + (-1.0)*tile[threadIdx.y-2][threadIdx.x+2]
				+ (-2.0)*tile[threadIdx.y-1][threadIdx.x-2] + (-8.0)*tile[threadIdx.y-1][threadIdx.x-1] + (-12.0)*tile[threadIdx.y-1][threadIdx.x] + (-8.0)*tile[threadIdx.y-1][threadIdx.x+1] + (-2.0)*tile[threadIdx.y-1][threadIdx.x+2]
				+    2.0*tile[threadIdx.y+1][threadIdx.x-2] +    8.0*tile[threadIdx.y+1][threadIdx.x-1] +    12.0*tile[threadIdx.y+1][threadIdx.x] +    8.0*tile[threadIdx.y+1][threadIdx.x+1] +    2.0*tile[threadIdx.y+1][threadIdx.x+2]
				+    1.0*tile[threadIdx.y+2][threadIdx.x-2] +    4.0*tile[threadIdx.y+2][threadIdx.x-1] +     6.0*tile[threadIdx.y+2][threadIdx.x] +    4.0*tile[threadIdx.y+2][threadIdx.x+1] +    1.0*tile[threadIdx.y+2][threadIdx.x+2]);

			G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
			phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

			if(fabs(phi[i*width+j])<=PI/8 )
				phi[i*width+j] = 0;
			else if (fabs(phi[i*width+j])<= 3*(PI/8))
				phi[i*width+j] = 45;
			else if (fabs(phi[i*width+j]) <= 5*(PI/8))
				phi[i*width+j] = 90;
			else if (fabs(phi[i*width+j]) <= 7*(PI/8))
				phi[i*width+j] = 135;
			else phi[i*width+j] = 0;
		}
	}
}

__global__ void gpu_canny_edge(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	__shared__ float tile [ TILE_DIM ] [ TILE_DIM ]; 
	i = blockIdx.y * NTHREADS + threadIdx.y; 
	j = blockIdx.x * NTHREADS + threadIdx.x; 
	if(i<height && j<width){
		tile[threadIdx.y][threadIdx.x] = G[i*width+j];
		__syncthreads();

		if(i>=3 && i<height-3 && threadIdx.x>=1 && threadIdx.x<TILE_DIM-3 && j >=3 && j<width-3 && threadIdx.y>=1 && threadIdx.y<TILE_DIM-3){
			pedge[i*width+j] = 0;
			if(phi[i*width+j] == 0){
				if(tile[i][j]>tile[i][j+1] && tile[i][j]>tile[i][j-1]) //edge is in N-S
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 45) {
				if(tile[i][j]>tile[i+1][j+1] && tile[i][j]>tile[i-1][j-1]) // edge is in NW-SE
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 90) {
				if(tile[i][j]>tile[i][j] && tile[i][j]>tile[i-1][j]) //edge is in E-W
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 135) {
				if(tile[i][j]>tile[i+1][j-1] && tile[i][j]>tile[i-1][j+1]) // edge is in NE-SW
					pedge[i*width+j] = 1;
			}
		}
	}
}

__global__ void gpu_canny_out(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	int ii, jj;
	float lowthres, hithres;
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
	// Hysteresis Thresholding
	lowthres = level/2;
	hithres  = 2*(level);
	if(i>=3 && i<height-3 && j >=3 && j<width-3){
		image_out[i*width+j] = 0;
			if(G[i*width+j]>hithres && pedge[i*width+j])
				image_out[i*width+j] = 255;
			else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
				// check neighbours 3x3
				for (ii=-1;ii<=1; ii++)
					for (jj=-1;jj<=1; jj++)
						if (G[(i+ii)*width+j+jj]>hithres)
							image_out[i*width+j] = 255;
	}
}


__global__ void gpu_hough_init(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, 
	float *sin_table, float *cos_table)
{
	int j;
	j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j<(accu_width*accu_height)){
		accumulators[j]=0;
	}
}

__global__ void gpu_hough_exec(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, 
	float *sin_table, float *cos_table)
{
	int i, j, theta;
	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);
	float center_x = width/2.0; 
	float center_y = height/2.0;
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
	if(i<height && j<width){
		if( im[ (i*width) + j] > 250 ) // Pixel is edge  
			{  
				for(theta=0;theta<180;theta++)  
				{  
					float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
					accumulators[ (int)((round(rho + hough_h) * 180.0)) + theta]++;

				} 
			} 
	}
}

void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table,
	int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho, theta, ii, jj;
	uint32_t max;

	for(rho=0;rho<accu_height;rho++)
	{
		for(theta=0;theta<accu_width;theta++)  
		{  

			if(accumulators[(rho*accu_width) + theta] >= threshold)  
			{  
				//Is this point a local maxima (9x9)  
				max = accumulators[(rho*accu_width) + theta]; 
				for(int ii=-4;ii<=4;ii++)  
				{  
					for(int jj=-4;jj<=4;jj++)  
					{  
						if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) )  
						{  
							if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max )  
							{
								max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
							}  
						}  
					}  
				}  

				if(max == accumulators[(rho*accu_width) + theta]) //local maxima
				{
					int x1, y1, x2, y2;  
					x1 = y1 = x2 = y2 = 0;  

					if(theta >= 45 && theta <= 135)  
					{
						if (theta>90) {
							//y = (r - x cos(t)) / sin(t)  
							x1 = width/2;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						} else {
							//y = (r - x cos(t)) / sin(t)  
							x1 = 0;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width*2/5;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					} else {
						//x = (r - y sin(t)) / cos(t);  
						y1 = 0;  
						x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2 = height;  
						x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}
					x1_lines[*lines] = x1;
					y1_lines[*lines] = y1;
					x2_lines[*lines] = x2;
					y2_lines[*lines] = y2;
					(*lines)++;
				}
			}
		}
	}
}

void canny(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level,
	int height, int width)
{
	int i, j;
	int ii, jj;
	float PI = 3.141593;

	float lowthres, hithres;

	for(i=2; i<height-2; i++)
		for(j=2; j<width-2; j++)
		{
			// Noise reduction
			NR[i*width+j] =
				 (2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
				+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
				+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
				+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
				+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
				/159.0;
		}


	for(i=2; i<height-2; i++)
		for(j=2; j<width-2; j++)
		{
			// Intensity gradient of the image
			Gx[i*width+j] = 
				 (1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
				+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
				+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
				+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


			Gy[i*width+j] = 
				 ((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
				+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
				+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);

			G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
			phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

			if(fabs(phi[i*width+j])<=PI/8 )
				phi[i*width+j] = 0;
			else if (fabs(phi[i*width+j])<= 3*(PI/8))
				phi[i*width+j] = 45;
			else if (fabs(phi[i*width+j]) <= 5*(PI/8))
				phi[i*width+j] = 90;
			else if (fabs(phi[i*width+j]) <= 7*(PI/8))
				phi[i*width+j] = 135;
			else phi[i*width+j] = 0;
	}

	// Edge
	for(i=3; i<height-3; i++)
		for(j=3; j<width-3; j++)
		{
			pedge[i*width+j] = 0;
			if(phi[i*width+j] == 0){
				if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 45) {
				if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 90) {
				if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 135) {
				if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
					pedge[i*width+j] = 1;
			}
		}

	// Hysteresis Thresholding
	lowthres = level/2;
	hithres  = 2*(level);

	for(i=3; i<height-3; i++)
		for(j=3; j<width-3; j++)
		{
			image_out[i*width+j] = 0;
			if(G[i*width+j]>hithres && pedge[i*width+j])
				image_out[i*width+j] = 255;
			else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
				// check neighbours 3x3
				for (ii=-1;ii<=1; ii++)
					for (jj=-1;jj<=1; jj++)
						if (G[(i+ii)*width+j+jj]>hithres)
							image_out[i*width+j] = 255;
		}
}

void line_asist_GPU(uint8_t *im, int height, int width,
	uint8_t *imEdge, float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float *sin_table, float *cos_table, 
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *x2, int *y1, int *y2, int *nlines)
{
	int threshold;

	// Create temporal buffers 
	uint8_t *imEdge_CPU = (uint8_t *)malloc(sizeof(uint8_t) * width * height);
	float *NR_CPU = (float *)malloc(sizeof(float) * width * height);
	float *G_CPU = (float *)malloc(sizeof(float) * width * height);
	float *phi_CPU = (float *)malloc(sizeof(float) * width * height);
	float *Gx_CPU = (float *)malloc(sizeof(float) * width * height);
	float *Gy_CPU = (float *)malloc(sizeof(float) * width * height);
	uint8_t *pedge_CPU = (uint8_t *)malloc(sizeof(uint8_t) * width * height);

	canny(im, imEdge_CPU,
		NR_CPU, G_CPU, phi_CPU, Gx_CPU, Gy_CPU, pedge_CPU,
		1000.0f, //level
		height, width);


	/* CUDA vesion */

	dim3 dimBlock(TILE_DIM,TILE_DIM);
	int blocks_h = (height)/NTHREADS;
	int blocks_w = (width)/NTHREADS;
	if ((height)%NTHREADS>0) blocks_h++;
	if ((width)%NTHREADS>0) blocks_w++;
	dim3 dimGrid(blocks_w, blocks_h);

	gpu_canny_nr<<<dimGrid,dimBlock>>>(im, imEdge, NR, G, phi, Gx, Gy, pedge,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();
	gpu_canny_gradient<<<dimGrid,dimBlock>>>(im, imEdge, NR, G, phi, Gx, Gy, pedge,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();
	gpu_canny_edge<<<dimGrid,dimBlock>>>(im, imEdge, NR, G, phi, Gx, Gy, pedge,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();

	dim3 dimBlock_out(NTHREADS, NTHREADS);
	gpu_canny_out<<<dimGrid,dimBlock_out>>>(im, imEdge, NR, G, phi, Gx, Gy, pedge,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();

	//Comprobacion de canny
	int i, j, desigual;
	i=0;
	desigual=0;
	while(i<height && !desigual){
		j=0;
		while(j<width && !desigual){
			if(NR[i*width+j] != NR_CPU[i*width+j]){
				printf("NR!= : NR_GPU[%d][%d]: %f  NR_GPU[%d][%d]: %f\n", i, j, NR[i*width+j], i,j,NR_CPU[i*width+j]);
				desigual=1;
			}
			if(G[i*width+j] != G_CPU[i*width+j]){
				printf("G!= : G_GPU[%d][%d]: %f  G_GPU[%d][%d]: %f\n", i, j, G[i*width+j], i,j,G_CPU[i*width+j]);
				desigual=1;
			}
			j++;
		}
		i++;
	}

	dim3 dimBlock_init(NTHREADS);
	blocks_h = (accu_height*accu_width)/NTHREADS;
	if ((accu_height*accu_width)%NTHREADS>0) blocks_h++;
	dim3 dimGrid_init(blocks_h);
	gpu_hough_init<<<dimGrid_init,dimBlock_init>>>(imEdge, width, height, accum, accu_width, accu_height, 
		sin_table, cos_table);
	hipDeviceSynchronize();

	dim3 dimBlock_exec(NTHREADS, NTHREADS);
	blocks_h = (height)/NTHREADS;
	blocks_w = (width)/NTHREADS;
	if ((height)%NTHREADS>0) blocks_h++;
	if ((width)%NTHREADS>0) blocks_w++;
	dim3 dimGrid_exec(blocks_w, blocks_h);
	gpu_hough_exec<<<dimGrid_exec,dimBlock_exec>>>(imEdge, width, height, accum, accu_width, accu_height, 
		sin_table, cos_table);
	hipDeviceSynchronize();

	if (width>height) threshold = width/6;
	else threshold = height/6;
	
	getlines(threshold, accum, accu_width, accu_height, width, height, 
		sin_table, cos_table,
		x1, y1, x2, y2, nlines);
}