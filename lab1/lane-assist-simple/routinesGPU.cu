#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"

#define NTHREADS 16

__global__ void gpu_canny_nr(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
	if(i>=2 && i<height-2 && j >=2 && j<width-2){
		// Noise reduction
		NR[i*width+j] =
				(2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
			+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
			+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
			+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
			+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
			/159.0;
	}

}

__global__ void gpu_canny_gradient(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	float PI = 3.141593;
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
	if(i<height && j<width){
		if(i>=2 && i<height-2 && j >=2 && j<width-2){
			// Intensity gradient of the image
			Gx[i*width+j] = 
				 (1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
				+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
				+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
				+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


			Gy[i*width+j] = 
				 ((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
				+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
				+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);

			G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
			phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

			if(fabs(phi[i*width+j])<=PI/8 )
				phi[i*width+j] = 0;
			else if (fabs(phi[i*width+j])<= 3*(PI/8))
				phi[i*width+j] = 45;
			else if (fabs(phi[i*width+j]) <= 5*(PI/8))
				phi[i*width+j] = 90;
			else if (fabs(phi[i*width+j]) <= 7*(PI/8))
				phi[i*width+j] = 135;
			else phi[i*width+j] = 0;
		}
	}
}

__global__ void gpu_canny_edge(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
	if(i<height && j<width){
		if(i>=3 && i<height-3 && j >=3 && j<width-3){
			pedge[i*width+j] = 0;
			if(phi[i*width+j] == 0){
				if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 45) {
				if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 90) {
				if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 135) {
				if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
					pedge[i*width+j] = 1;
			}
		}
	}
}

__global__ void gpu_canny_out(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level, int height, int width)
{
	int i, j; 
	int ii, jj;
	float lowthres, hithres;
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
	// Hysteresis Thresholding
	lowthres = level/2;
	hithres  = 2*(level);
	if(i>=3 && i<height-3 && j >=3 && j<width-3){
		image_out[i*width+j] = 0;
			if(G[i*width+j]>hithres && pedge[i*width+j])
				image_out[i*width+j] = 255;
			else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
				// check neighbours 3x3
				for (ii=-1;ii<=1; ii++)
					for (jj=-1;jj<=1; jj++)
						if (G[(i+ii)*width+j+jj]>hithres)
							image_out[i*width+j] = 255;
	}
}


__global__ void gpu_hough_init(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, 
	float *sin_table, float *cos_table)
{
	int j;
	j = blockIdx.x * blockDim.x + threadIdx.x;
	if(j<(accu_width*accu_height)){
		accumulators[j]=0;
	}
}

__global__ void gpu_hough_exec(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, 
	float *sin_table, float *cos_table)
{
	int i, j, theta;
	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);
	float center_x = width/2.0; 
	float center_y = height/2.0;
	i = blockIdx.y * blockDim.y + threadIdx.y; 
	j = blockIdx.x * blockDim.x + threadIdx.x; 
	if(i<height && j<width){
		if( im[ (i*width) + j] > 250 ) // Pixel is edge  
			{  
				for(theta=0;theta<180;theta++)  
				{  
					float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
					accumulators[ (int)((round(rho + hough_h) * 180.0)) + theta]++;

				} 
			} 
	}
}

void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table,
	int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho, theta, ii, jj;
	uint32_t max;

	for(rho=0;rho<accu_height;rho++)
	{
		for(theta=0;theta<accu_width;theta++)  
		{  

			if(accumulators[(rho*accu_width) + theta] >= threshold)  
			{  
				//Is this point a local maxima (9x9)  
				max = accumulators[(rho*accu_width) + theta]; 
				for(int ii=-4;ii<=4;ii++)  
				{  
					for(int jj=-4;jj<=4;jj++)  
					{  
						if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) )  
						{  
							if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max )  
							{
								max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
							}  
						}  
					}  
				}  

				if(max == accumulators[(rho*accu_width) + theta]) //local maxima
				{
					int x1, y1, x2, y2;  
					x1 = y1 = x2 = y2 = 0;  

					if(theta >= 45 && theta <= 135)  
					{
						if (theta>90) {
							//y = (r - x cos(t)) / sin(t)  
							x1 = width/2;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						} else {
							//y = (r - x cos(t)) / sin(t)  
							x1 = 0;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width*2/5;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					} else {
						//x = (r - y sin(t)) / cos(t);  
						y1 = 0;  
						x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2 = height;  
						x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}
					x1_lines[*lines] = x1;
					y1_lines[*lines] = y1;
					x2_lines[*lines] = x2;
					y2_lines[*lines] = y2;
					(*lines)++;
				}
			}
		}
	}
}

// Para comprobar las imagenes
void canny(uint8_t *im, uint8_t *image_out,
	float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float level,
	int height, int width)
{
	int i, j;
	int ii, jj;
	float PI = 3.141593;

	float lowthres, hithres;

	for(i=2; i<height-2; i++)
		for(j=2; j<width-2; j++)
		{
			// Noise reduction
			NR[i*width+j] =
				 (2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
				+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
				+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
				+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
				+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
				/159.0;
		}


	for(i=2; i<height-2; i++)
		for(j=2; j<width-2; j++)
		{
			// Intensity gradient of the image
			Gx[i*width+j] = 
				 (1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
				+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
				+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
				+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


			Gy[i*width+j] = 
				 ((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
				+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
				+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);

			G[i*width+j]   = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));	//G = √Gx²+Gy²
			phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

			if(fabs(phi[i*width+j])<=PI/8 )
				phi[i*width+j] = 0;
			else if (fabs(phi[i*width+j])<= 3*(PI/8))
				phi[i*width+j] = 45;
			else if (fabs(phi[i*width+j]) <= 5*(PI/8))
				phi[i*width+j] = 90;
			else if (fabs(phi[i*width+j]) <= 7*(PI/8))
				phi[i*width+j] = 135;
			else phi[i*width+j] = 0;
	}

	// Edge
	for(i=3; i<height-3; i++)
		for(j=3; j<width-3; j++)
		{
			pedge[i*width+j] = 0;
			if(phi[i*width+j] == 0){
				if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 45) {
				if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 90) {
				if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
					pedge[i*width+j] = 1;

			} else if(phi[i*width+j] == 135) {
				if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
					pedge[i*width+j] = 1;
			}
		}

	// Hysteresis Thresholding
	lowthres = level/2;
	hithres  = 2*(level);

	for(i=3; i<height-3; i++)
		for(j=3; j<width-3; j++)
		{
			image_out[i*width+j] = 0;
			if(G[i*width+j]>hithres && pedge[i*width+j])
				image_out[i*width+j] = 255;
			else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
				// check neighbours 3x3
				for (ii=-1;ii<=1; ii++)
					for (jj=-1;jj<=1; jj++)
						if (G[(i+ii)*width+j+jj]>hithres)
							image_out[i*width+j] = 255;
		}
}

//Para comprobar hough
void houghtransform(uint8_t *im, int width, int height, uint32_t *accumulators, int accu_width, int accu_height, 
	float *sin_table, float *cos_table)
{
	int i, j, theta;

	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);

	for(i=0; i<accu_width*accu_height; i++)
		accumulators[i]=0;	

	float center_x = width/2.0; 
	float center_y = height/2.0;
	for(i=0;i<height;i++)  
	{  
		for(j=0;j<width;j++)  
		{  
			if( im[ (i*width) + j] > 250 ) // Pixel is edge  
			{  
				for(theta=0;theta<180;theta++)  
				{  
					float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
					accumulators[ (int)((round(rho + hough_h) * 180.0)) + theta]++;

				} 
			} 
		} 
	}
}

void line_asist_GPU(uint8_t *im, int height, int width,
	uint8_t *imEdge, float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float *sin_table, float *cos_table, 
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *x2, int *y1, int *y2, int *nlines)
{
	int threshold;

	// Create temporal buffers 
	uint8_t *imEdge_CPU = (uint8_t *)malloc(sizeof(uint8_t) * width * height);
	float *NR_CPU = (float *)malloc(sizeof(float) * width * height);
	float *G_CPU = (float *)malloc(sizeof(float) * width * height);
	float *phi_CPU = (float *)malloc(sizeof(float) * width * height);
	float *Gx_CPU = (float *)malloc(sizeof(float) * width * height);
	float *Gy_CPU = (float *)malloc(sizeof(float) * width * height);
	uint8_t *pedge_CPU = (uint8_t *)malloc(sizeof(uint8_t) * width * height);

	
	uint32_t *accum_CPU = (uint32_t*)malloc(accu_width*accu_height*sizeof(uint32_t));

	canny(im, imEdge_CPU,
		NR_CPU, G_CPU, phi_CPU, Gx_CPU, Gy_CPU, pedge_CPU,
		1000.0f, //level
		height, width);

	/* hough transform */
	houghtransform(imEdge, width, height, accum_CPU, accu_width, accu_height, sin_table, cos_table);


	/* CUDA vesion */

	uint8_t *im_GPU;
	uint8_t *imEdge_GPU;
	float *NR_GPU;
	float *G_GPU;
	float *phi_GPU; 
	float *Gx_GPU;
	float *Gy_GPU;
	uint8_t *pedge_GPU; 
	int  size_float = sizeof(float) * width * height;
	int  size_uint8 = sizeof(uint8_t) * width * height;
	hipMalloc((void **)&im_GPU, size_uint8);
	hipMalloc((void **)&imEdge_GPU, size_uint8);
	hipMalloc((void **)&NR_GPU, size_float);
	hipMalloc((void **)&G_GPU, size_float);
	hipMalloc((void **)&phi_GPU, size_float);
	hipMalloc((void **)&Gx_GPU, size_float);
	hipMalloc((void **)&Gy_GPU, size_float);
	hipMalloc((void **)&pedge_GPU, size_uint8);

	hipMemcpy(im_GPU, im, size_uint8, hipMemcpyHostToDevice);

	dim3 dimBlock(NTHREADS,NTHREADS);
	int blocks_h = (height)/NTHREADS;
	int blocks_w = (width)/NTHREADS;
	if ((height)%NTHREADS>0) blocks_h++;
	if ((width)%NTHREADS>0) blocks_w++;
	dim3 dimGrid(blocks_w, blocks_h);

	gpu_canny_nr<<<dimGrid,dimBlock>>>(im_GPU, imEdge_GPU, NR_GPU, G_GPU, phi_GPU, Gx_GPU, Gy_GPU, pedge_GPU,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();
	
	gpu_canny_gradient<<<dimGrid,dimBlock>>>(im_GPU, imEdge_GPU, NR_GPU, G_GPU, phi_GPU, Gx_GPU, Gy_GPU, pedge_GPU,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();
	gpu_canny_edge<<<dimGrid,dimBlock>>>(im_GPU, imEdge_GPU, NR_GPU, G_GPU, phi_GPU, Gx_GPU, Gy_GPU, pedge_GPU,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();

	dim3 dimBlock_out(NTHREADS, NTHREADS);
	gpu_canny_out<<<dimGrid,dimBlock_out>>>(im_GPU, imEdge_GPU, NR_GPU, G_GPU, phi_GPU, Gx_GPU, Gy_GPU, pedge_GPU,
		1000.0f, //level
		height, width);	
	hipDeviceSynchronize();

	uint32_t *accum_GPU;
	float* sin_table_GPU;
	float* cos_table_GPU;
	int size_accum = accu_width*accu_height*sizeof(uint32_t);
	int size_table = 180*sizeof(float);
	hipMalloc((void **)&accum_GPU, size_accum);
	hipMalloc((void **)&sin_table_GPU, size_table);
	hipMalloc((void **)&cos_table_GPU, size_table);

	hipMemcpy(sin_table_GPU, sin_table, size_table, hipMemcpyHostToDevice);
	hipMemcpy(cos_table_GPU, cos_table, size_table, hipMemcpyHostToDevice);

	dim3 dimBlock_init(NTHREADS);
	blocks_h = (accu_height*accu_width)/NTHREADS;
	if ((accu_height*accu_width)%NTHREADS>0) blocks_h++;
	dim3 dimGrid_init(blocks_h);
	gpu_hough_init<<<dimGrid_init,dimBlock_init>>>(imEdge_GPU, width, height, accum_GPU, accu_width, accu_height, 
		sin_table_GPU, cos_table_GPU);
	hipDeviceSynchronize();

	gpu_hough_exec<<<dimGrid,dimBlock>>>(imEdge_GPU, width, height, accum_GPU, accu_width, accu_height, 
		sin_table_GPU, cos_table_GPU);
	hipDeviceSynchronize();

	hipMemcpy(imEdge, imEdge_GPU, size_uint8, hipMemcpyDeviceToHost);
	hipMemcpy(NR, NR_GPU, size_float, hipMemcpyDeviceToHost);
	hipMemcpy(G, G_GPU, size_float, hipMemcpyDeviceToHost);
	hipMemcpy(phi, phi_GPU, size_float, hipMemcpyDeviceToHost);
	hipMemcpy(Gx, Gx_GPU, size_float, hipMemcpyDeviceToHost);
	hipMemcpy(Gy, Gy_GPU, size_float, hipMemcpyDeviceToHost);
	hipMemcpy(pedge, pedge_GPU, size_uint8, hipMemcpyDeviceToHost);
	hipMemcpy(accum, accum_GPU, size_accum, hipMemcpyDeviceToHost);

	hipFree(im_GPU);
	hipFree(imEdge_GPU);
	hipFree(NR_GPU);
	hipFree(G_GPU);
	hipFree(phi_GPU); 
	hipFree(Gx_GPU);
	hipFree(Gy_GPU);
	hipFree(pedge_GPU); 
	hipFree(accum_GPU); 
	hipFree(sin_table_GPU);
	hipFree(cos_table_GPU);

	//Comprobacion de canny
	int i, j, desigual;
	i=0;
	desigual=0;
	while(i<height && desigual<5){
		j=0;
		while(j<width && desigual<5){
			/*if(NR[i*width+j] != NR_CPU[i*width+j]){
				printf("NR!= : NR_GPU[%d][%d]: %f  NR_CPU[%d][%d]: %f\n", i, j, NR[i*width+j], i,j,NR_CPU[i*width+j]);
				desigual++;
			} BIEN */
			/*if(G[i*width+j] != G_CPU[i*width+j]){
				printf("G!= : G_GPU[%d][%d]: %f  G_CPU[%d][%d]: %f\n", i, j, G[i*width+j], i,j,G_CPU[i*width+j]);
				desigual++;
			} POCA DIFERENCIA*/
			/*if(pedge[i*width+j] != pedge_CPU[i*width+j]){
				printf("pedge!= : pedge_GPU[%d][%d]: %d  pedge_CPU[%d][%d]: %d\n", i, j, pedge[i*width+j], i,j,pedge_CPU[i*width+j]);
				desigual++;
			}*/
			/*if(phi[i*width+j] != phi_CPU[i*width+j]){
				printf("phi!= : phi_GPU[%d][%d]: %f  phi_CPU[%d][%d]: %f\n", i, j, phi[i*width+j], i,j,phi_CPU[i*width+j]);
				desigual=1;
			} BIEN */
			/*if(accum[i*width+j] != accum_CPU[i*width+j]){
				printf("accum!= : accum_GPU[%d][%d]: %d  accum_CPU[%d][%d]: %d\n", i, j, accum[i*width+j], i,j,accum_CPU[i*width+j]);
				desigual=1;
			}*/
			j++;
		}
		i++;
	}

	if (width>height) threshold = width/6;
	else threshold = height/6;
	
	getlines(threshold, accum, accu_width, accu_height, width, height, 
		sin_table, cos_table,
		x1, y1, x2, y2, nlines);
}